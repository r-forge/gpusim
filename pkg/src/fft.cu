
#include <stdio.h>
#include <stdlib.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>




#define MAX_PLANS 10



typedef struct {
  int nx;
  int ny;
  int nz;
  int use_double;
  hipfftHandle cufftPlan;
} fft_plan;



fft_plan *plans[MAX_PLANS];
int cur_plan_index;



#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

// C oder C++ Compiler
#ifdef __cplusplus
extern "C" {
#endif

//alle Funktionen, die mit EXPORT gekennzeichnet sind, koennen in R ueber .C aufgerufen werden
void EXPORT initFFT() {

	for (int i=0; i<MAX_PLANS; ++i) {
		plans[i] = 0; // init plans with null pointers
	}
	cur_plan_index = 0;
}



void EXPORT planFFT(int *nx, int *ny, int *nz, int* use_double) {
  fft_plan *plan = (fft_plan*)malloc(sizeof(fft_plan));
  plan->nx = *nx; //*plan.nx = nx;
  plan->ny = *ny;
  plan->nz = *nz;
  plan->use_double = *use_double;

  //plan->h_data  = (cufftComplex*)malloc(sizeof(cufftComplex) * plan->nx * plan->ny * plan->nz);
  //cudaMalloc((void**)&plan->d_data,sizeof(cufftComplex) * plan->nx * plan->ny * plan->nz);
  
  hipfftType type = (*use_double)? HIPFFT_Z2Z : HIPFFT_C2C;
  // if (*use_double) type=CUFFT_Z2Z else type=CUFFT_C2C

  // Plan je nach Dimension
  if (plan->nz == 1 && plan->ny == 1) {
	hipfftPlan1d(&plan->cufftPlan, plan->nx, type, 1);
  }
  else if (plan->nz == 1) {
	hipfftPlan2d(&plan->cufftPlan, plan->ny, plan->nx, type); // cufft uses row-major
  }
  else {
	hipfftPlan3d(&plan->cufftPlan, plan->nz, plan->ny, plan->nx, type); // cufft uses row-major
  }
  
  if (plans[cur_plan_index] != 0) {
	  hipfftDestroy(plans[cur_plan_index]->cufftPlan);
	  free(plans[cur_plan_index]);
	  plans[cur_plan_index] = 0;
  }
  plans[cur_plan_index] = plan;
  cur_plan_index = (cur_plan_index + 1) % MAX_PLANS;
}




void EXPORT execFFT(double *out, double *data, int *nx, int *ny, int *nz, int *inverse, int *is_complex, int *use_double) {

	// Is there already a plan?!
	fft_plan *plan = 0;
	for (int i=0; i<MAX_PLANS; ++i) {
		if (plans[i] != 0) {
			if (plans[i]->nx == *nx && plans[i]->ny == *ny && plans[i]->nz == *nz && plans[i]->use_double == *use_double) {
				plan = plans[i];
				break;
			}
		}
	}
	// If not, create one
	if (plan == 0) {
		planFFT(nx,ny,nz,use_double);
		plan = plans[(cur_plan_index - 1) % MAX_PLANS];
	}


	int direction = HIPFFT_FORWARD;
	if (*inverse) direction = HIPFFT_BACKWARD;  
  
	int n = plan->nx * plan->ny * plan->nz;
	
	
	if (*use_double) {
		hipfftDoubleComplex *h_data = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * n);
		hipfftDoubleComplex *d_data;
		hipMalloc((void**)&d_data,sizeof(hipfftDoubleComplex) * n);

		if (!*is_complex) {
			for (int i = 0; i < n; ++i) {
			  h_data[i].x = (double)data[i];
			  h_data[i].y = 0.0;
			}
		}
		else {
			for (int i = 0; i < n; ++i) {
			  h_data[i].x = (double)data[2*i];
			  h_data[i].y = (double)data[2*i+1];
			}

		}
		// Copy data to GPU memory
		hipMemcpy(d_data,h_data,sizeof(hipfftDoubleComplex) * n, hipMemcpyHostToDevice);
		// Execute FFT in place
		hipfftExecZ2Z(plan->cufftPlan,d_data,d_data,direction);
		// Copy data back to host memory
		hipMemcpy(h_data,d_data,sizeof(hipfftDoubleComplex) * n, hipMemcpyDeviceToHost);	

        // komplexer Datentyp von CUDA wird in 2*n Array umgewandelt
		for (int i = 0; i < n; ++i) {
			out[2*i] = h_data[i].x;
			out[2*i+1] = h_data[i].y;
		}
		free(h_data);
		hipFree(d_data);
	}
    // das gleiche fuer float
	else {
		hipfftComplex *h_data = (hipfftComplex*)malloc(sizeof(hipfftComplex) * n);
		hipfftComplex *d_data;
		hipMalloc((void**)&d_data,sizeof(hipfftComplex) * n);

		if (!*is_complex) {
			for (int i = 0; i < n; ++i) {
			  h_data[i].x = (float)data[i];
			  h_data[i].y = 0.0f;
			}
		}
		else {
			for (int i = 0; i < n; ++i) {
			  h_data[i].x = (float)data[2*i];
			  h_data[i].y = (float)data[2*i+1];
			}

		}
		// Copy data to GPU memory
		hipMemcpy(d_data,h_data,sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
		// Execute FFT in place
		hipfftExecC2C(plan->cufftPlan,d_data,d_data,direction);
		// Copy data back to host memory
		hipMemcpy(h_data,d_data,sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);	


		for (int i = 0; i < n; ++i) {
			out[2*i] = h_data[i].x;
			out[2*i+1] = h_data[i].y;
		}
		free(h_data);
		hipFree(d_data);
	}
}






#ifdef __cplusplus
}
#endif
