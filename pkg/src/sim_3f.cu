#include "hip/hip_runtime.h"
/**
* sim.cu C and CUDA Interface for gpusim R package
* Author: Marius Appel - marius.appel@uni-muenster.de
*
* TODO: 
*	- Split into several files
*	- Add functions to minimize redundant code
*	- introduce debug args for simpler bug finding
*
*	14.02.2012
**/

#include "utils.h"



/*******************************************************************************************
** GPU KERNELS *****************************************************************************
********************************************************************************************/

__device__ float covExpKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget) {
	float dist = sqrt((ax-bx)*(ax-bx)+(ay-by)*(ay-by)+(az-bz)*(az-bz));
	return ((dist == 0.0f)? (nugget + sill) : (sill*exp(-dist/range)));
}


__device__ float covExpAnisKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2) {
	float dist = 0.0;
	float temp = 0.0;
	float dx = ax-bx;
	float dy = ay-by;
	float dz = az-bz;	
	temp = dx*cos(beta)*cos(alpha) + dy*cos(beta)*sin(alpha) - dz * sin(beta);
	dist += temp * temp;
	temp = afac1 * (-dx * (cos(theta)*sin(alpha) + sin(theta)*sin(beta)*cos(alpha)) + 
						dy * (cos(theta)*cos(alpha) + sin(theta)*sin(beta)*sin(alpha)) + 
						dz * sin(theta)*cos(beta));
	dist += temp * temp;
	temp = afac2 * (dx * (sin(theta)*sin(alpha) + cos(theta)*sin(beta)*cos(alpha)) + 
					dy * (-sin(theta)*cos(alpha) + cos(theta)*sin(beta)*sin(alpha)) + 
					dz * cos(theta) * cos(beta));		
	dist += temp * temp;
	dist = sqrt(dist);
	return ((dist == 0.0f)? (nugget + sill) : (sill*exp(-dist/range)));
}



__device__ float covGauKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget) {
	float dist2 = (ax-bx)*(ax-bx)+(ay-by)*(ay-by)+(az-bz)*(az-bz);
	return ((dist2 == 0.0f)? (nugget + sill) : (sill*exp(-dist2/(range*range))));
}



__device__ float covGauAnisKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2) {
	
	float dist = 0.0;
	float temp = 0.0;
	float dx = ax-bx;
	float dy = ay-by;
	float dz = az-bz;	
	temp = dx*cos(beta)*cos(alpha) + dy*cos(beta)*sin(alpha) - dz * sin(beta);
	dist += temp * temp;
	temp = afac1 * (-dx * (cos(theta)*sin(alpha) + sin(theta)*sin(beta)*cos(alpha)) + 
						dy * (cos(theta)*cos(alpha) + sin(theta)*sin(beta)*sin(alpha)) + 
						dz * sin(theta)*cos(beta));
	dist += temp * temp;
	temp = afac2 * (dx * (sin(theta)*sin(alpha) + cos(theta)*sin(beta)*cos(alpha)) + 
					dy * (-sin(theta)*cos(alpha) + cos(theta)*sin(beta)*sin(alpha)) + 
					dz * cos(theta) * cos(beta));		
	dist += temp * temp;
	//dist = sqrt(dist);
	return ((dist == 0.0f)? (nugget + sill) : (sill*exp(-dist/(range*range))));
}





__device__ float covSphKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget) {
	float dist = sqrt((ax-bx)*(ax-bx)+(ay-by)*(ay-by)+(az-bz)*(az-bz));
	if (dist == 0.0) 
		return(nugget + sill);	
	else if(dist <= range) 
		return sill * (1.0 - (((3.0*dist) / (2.0*range)) - ((dist * dist * dist) / (2.0 * range * range * range)) ));	
	return 0.0f; // WARNING,  sample cov matrix may be not regular for wenn point pairs with distance > range
}




__device__ float covSphAnisKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2) {
	float dist = 0.0;
	float temp = 0.0;
	float dx = ax-bx;
	float dy = ay-by;
	float dz = az-bz;	
	temp = dx*cos(beta)*cos(alpha) + dy*cos(beta)*sin(alpha) - dz * sin(beta);
	dist += temp * temp;
	temp = afac1 * (-dx * (cos(theta)*sin(alpha) + sin(theta)*sin(beta)*cos(alpha)) + 
						dy * (cos(theta)*cos(alpha) + sin(theta)*sin(beta)*sin(alpha)) + 
						dz * sin(theta)*cos(beta));
	dist += temp * temp;
	temp = afac2 * (dx * (sin(theta)*sin(alpha) + cos(theta)*sin(beta)*cos(alpha)) + 
					dy * (-sin(theta)*cos(alpha) + cos(theta)*sin(beta)*sin(alpha)) + 
					dz * cos(theta) * cos(beta));		
	dist += temp * temp;
	dist = sqrt(dist);
	if (dist == 0.0) 
		return(nugget + sill);	
	else if(dist <= range) 
		return sill * (1.0 - (((3.0*dist) / (2.0*range)) - ((dist * dist * dist) / (2.0 * range * range * range)) ));	
	return 0.0f; // WARNING,  sample cov matrix may be not regular for wenn point pairs with distance > range
}












__device__ float covMat3Kernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget) {
	float dist = sqrtf((ax-bx)*(ax-bx)+(ay-by)*(ay-by)+(az-bz)*(az-bz));
	return ((dist == 0.0f)? (nugget + sill) : (sill*(1+SQRT3*dist/range)*exp(-SQRT3*dist/range)));
}


__device__ float covMat3AnisKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2) {
	float dist = 0.0;
	float temp = 0.0;
	float dx = ax-bx;
	float dy = ay-by;
	float dz = az-bz;	
	temp = dx*cos(beta)*cos(alpha) + dy*cos(beta)*sin(alpha) - dz * sin(beta);
	dist += temp * temp;
	temp = afac1 * (-dx * (cos(theta)*sin(alpha) + sin(theta)*sin(beta)*cos(alpha)) + 
						dy * (cos(theta)*cos(alpha) + sin(theta)*sin(beta)*sin(alpha)) + 
						dz * sin(theta)*cos(beta));
	dist += temp * temp;
	temp = afac2 * (dx * (sin(theta)*sin(alpha) + cos(theta)*sin(beta)*cos(alpha)) + 
					dy * (-sin(theta)*cos(alpha) + cos(theta)*sin(beta)*sin(alpha)) + 
					dz * cos(theta) * cos(beta));		
	dist += temp * temp;
	dist = sqrt(dist);
	return ((dist == 0.0f)? (nugget + sill) : (sill*(1+SQRT3*dist/range)*exp(-SQRT3*dist/range)));
}




__device__ float covMat5Kernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget) {
	float dist = sqrtf((ax-bx)*(ax-bx)+(ay-by)*(ay-by)+(az-bz)*(az-bz));
	return ((dist == 0.0f)? (nugget + sill) : (sill * (1 + SQRT5*dist/range + 5*dist*dist/3*range*range) * exp(-SQRT5*dist/range)));
}
__device__ float covMat5AnisKernel_3f(float ax, float ay, float az, float bx, float by, float bz, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2) {
	float dist = 0.0;
	float temp = 0.0;
	float dx = ax-bx;
	float dy = ay-by;
	float dz = az-bz;	
	temp = dx*cos(beta)*cos(alpha) + dy*cos(beta)*sin(alpha) - dz * sin(beta);
	dist += temp * temp;
	temp = afac1 * (-dx * (cos(theta)*sin(alpha) + sin(theta)*sin(beta)*cos(alpha)) + 
						dy * (cos(theta)*cos(alpha) + sin(theta)*sin(beta)*sin(alpha)) + 
						dz * sin(theta)*cos(beta));
	dist += temp * temp;
	temp = afac2 * (dx * (sin(theta)*sin(alpha) + cos(theta)*sin(beta)*cos(alpha)) + 
					dy * (-sin(theta)*cos(alpha) + cos(theta)*sin(beta)*sin(alpha)) + 
					dz * cos(theta) * cos(beta));		
	dist += temp * temp;
	dist = sqrt(dist);
	return ((dist == 0.0f)? (nugget + sill) : (sill * (1 + SQRT5*dist/range + 5*dist*dist/3*range*range) * exp(-SQRT5*dist/range)));
}


// Converts real float array into hipfftComplex array
__global__ void realToComplexKernel_3f(hipfftComplex *c, float* r, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n) {
		c[i].x = r[i];
		c[i].y = 0.0f;
	}
}

// TODO: CHECK CORECTNESS
__global__ void ReDiv_3f(float *out, hipfftComplex *c, float div,int nx, int ny, int nz) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int zdim = threadIdx.z + blockIdx.z * blockDim.z;
	if (col < nx && row < ny && zdim < nz) out[zdim*nx*ny + row*nx + col] = c[zdim*4*nx*ny +  row*2*nx + col].x / div; //////// !!!!!!
}




// Covariance sampling of a regular grid
__global__ void sampleCovKernel_3f(hipfftComplex *trickgrid,float3 *grid, hipfftComplex* cov, float xc, float yc, float zc, int model, float sill, float range, float nugget, int n, int m, int o) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int zdim = threadIdx.z + blockIdx.z * blockDim.z;
	if (col < n && row < m && zdim < o) {
		
		
		switch (model) {
		case EXP:
			cov[zdim*n*m + row*n + col].x = covExpKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget);
			break;
		case GAU:
			cov[zdim*n*m + row*n + col].x = covGauKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget);
			break;
		case SPH:
			cov[zdim*n*m + row*n + col].x = covSphKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget);
			break;
		case MAT3:
			cov[zdim*n*m + row*n + col].x = covMat3Kernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget);
			break;
		case MAT5:
			cov[zdim*n*m + row*n + col].x = covMat5Kernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget);
			break;
		}	
	

		cov[zdim*n*m + row*n + col].y = 0;	
		if (col == n/2-1 && row == m/2-1 && zdim == o/2-1) {
			trickgrid[zdim*n*m + row*n+col].x = 1.0f;
			trickgrid[zdim*n*m + row*n+col].y = 0.0f;
		}
		else {
			trickgrid[zdim*n*m + row*n+col].x = 0.0f;
			trickgrid[zdim*n*m + row*n+col].y = 0.0f;
		}
	}
}






__global__ void sampleCovAnisKernel_3f(hipfftComplex *trickgrid,float3 *grid, hipfftComplex* cov, float xc, float yc, float zc, int model, float sill, float range, float nugget, float alpha, float beta, float theta, float afac1, float afac2, int n, int m, int o) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int zdim = threadIdx.z + blockIdx.z * blockDim.z;
	if (col < n && row < m && zdim < o) {	
		switch (model) {
		case EXP:
			cov[zdim*n*m + row*n + col].x = covExpAnisKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget,alpha,beta,theta,afac1,afac2);
			break;
		case GAU:
			cov[zdim*n*m + row*n + col].x = covGauAnisKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget,alpha,beta,theta,afac1,afac2);
			break;
		case SPH:
			cov[zdim*n*m + row*n + col].x = covSphAnisKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget,alpha,beta,theta,afac1,afac2);
			break;
		case MAT3:
			cov[zdim*n*m + row*n + col].x = covMat3AnisKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget,alpha,beta,theta,afac1,afac2);
			break;
		case MAT5:
			cov[zdim*n*m + row*n + col].x = covMat5AnisKernel_3f(grid[zdim*n*m +row*n+col].x,grid[zdim*n*m + row*n+col].y,grid[zdim*n*m + row*n+col].z,xc,yc,zc,sill,range,nugget,alpha,beta,theta,afac1,afac2);
			break;
		}	
		cov[zdim*n*m + row*n + col].y = 0;	
		if (col == n/2-1 && row == m/2-1 && zdim == o/2-1) {
			trickgrid[zdim*n*m + row*n+col].x = 1.0f;
			trickgrid[zdim*n*m + row*n+col].y = 0.0f;
		}
		else {
			trickgrid[zdim*n*m + row*n+col].x = 0.0f;
			trickgrid[zdim*n*m + row*n+col].y = 0.0f;
		}
	}
}










// TODO: Calculate n*m*o before on cpu and use this value as one arg instead of 3!!!!
__global__ void multKernel_3f(hipfftComplex *fftgrid, int n, int m, int o) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	fftgrid[i].x = fftgrid[i].x*n*m*o;
	fftgrid[i].y = fftgrid[i].y*n*m*o;
}


// Devides spectral grid elementwise by fftgrid
__global__ void divideSpectrumKernel_3f(hipfftComplex *spectrum, hipfftComplex *fftgrid) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	float a = spectrum[i].x;
	float b = spectrum[i].y;
	float c = fftgrid[i].x;
	float d = fftgrid[i].y;
	spectrum[i].x = (a*c+b*d)/(c*c+d*d);
	spectrum[i].y = (b*c-a*d)/(c*c+d*d);
}



// Element-wise sqrt from spectral grid
__global__ void sqrtKernel_3f(hipfftComplex *spectrum) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	float re = spectrum[i].x;
	float im = spectrum[i].y;
	float sill = 0;
	float d = sqrt(re*re+im*im);
	float dsqrt = sqrt(d);
	if(re>0)
		sill = atan(im/re);
	if(re<0 && im>=0)
		sill = atan(im/re)+PI;
	if(re<0 && im<0)
		sill = atan(im/re)-PI;
	if(re==0 && im>0)
		sill = PI/2;
	if(re==0 && im<0)
		sill = -PI/2;
	spectrum[i].x = dsqrt*cos(sill/2);
	spectrum[i].y = dsqrt*sin(sill/2);
}



// Element-wise multiplication of two complex arrays
__global__ void elementProduct_3f(hipfftComplex *c, hipfftComplex *a, hipfftComplex *b, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n) {
		c[i].x = a[i].x * b[i].x - a[i].y * b[i].y;
		c[i].y = a[i].x * b[i].y + a[i].y * b[i].x;
	}
}


/// Kriging prediction at a regular grid with given samples for conditioning
#ifndef BLOCK_SIZE_KRIGE1
#define BLOCK_SIZE_KRIGE1 256
#endif



// TODO: KRIGING -> 3d

__global__ void addResSim_3f(float *res, float *uncond, int n) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) res[id] += uncond[id];
}

__global__ void addResSimMean_3f(float *res, float *uncond, int n, float mean) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) res[id] += uncond[id] + mean;
}


__global__ void overlay_3f(float3 *out, float3 *xy, float grid_minx, float grid_dx, float grid_maxy, float grid_dy, float grid_minz, float grid_dz, int numPoints) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < numPoints) {
		out[i].x = (xy[i].x - grid_minx)/grid_dx;
		out[i].y = (grid_maxy - grid_dy - xy[i].y)/grid_dy;
		out[i].z = (xy[i].z - grid_minz)/grid_dz;
	}
}


__global__ void residualsOrdinary_3f(float* res, float *srcdata, float *uncond_grid, float3 *indices, int nx, int ny, int nz, int numPoints) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numPoints) {
		
		// Trilinear interpolation
		float x = indices[id].x; 
		float y = indices[id].y;
		float z = indices[id].z;
		int row = floor(y); // y index of upper neighbour pixel
		int col = floor(x); // x index of lower neighbour pixel
		int zdim = floor(z); // z index of lower neighbour pixel
		x = (float)x - col; // Weight of right neighbour or 1 - weight of left neighbour
		y = (float)y - row; // Weight of lower neighbour or 1 - weight of upper neighbour
		z = (float)z - zdim;
		
		// Special cases
		if (col > nx-1) {
			x = 0.0f;col = nx-1;
		}
		else if (col < 0) {
			x = 0.0f;col=0;
		}
		if (row > nx-1) {
			y = 0.0f;row = nx-y;
		}	
		else if (row < 0) {
			y = 0.0f;row=0;
		}
		if (zdim > nz-1) {
			z = 0.0f;zdim = nz-1;
		}
		else if (zdim < 0) {
			z = 0.0f;zdim = 0;
		}

		float c00 = (1-y)*uncond_grid[zdim*nx*ny + row*nx + col]         +   y*uncond_grid[zdim*nx*ny + row*nx + col+1];
		float c10 = (1-y)*uncond_grid[(zdim+1)*nx*ny + row*nx + col]     +   y*uncond_grid[(zdim+1)*nx*ny + row*nx + col+1];
		float c01 = (1-y)*uncond_grid[zdim*nx*ny + (row+1)*nx + col]     +   y*uncond_grid[zdim*nx*ny + (row+1)*nx + col+1];
		float c11 = (1-y)*uncond_grid[(zdim+1)*nx*ny + (row+1)*nx + col] +   y*uncond_grid[(zdim+1)*nx*ny + (row+1)*nx + col+1];

		c00 = (1-z) * c00 + z*c10;
		c01 = (1-z) * c01 + z*c11;

		res[id] = srcdata[id] - ((1-x)*c00 + x*c01);
		
		/*res[id] = srcdata[id] - ((1-y) * ((1-x) * uncond_grid[row * nx + col] + x * uncond_grid[row * nx + col + 1]) + 
								  y * ((1-x) * uncond_grid[(row+1) * nx + col] + x * uncond_grid[(row+1) * nx + col + 1]));*/
	}		
	if (id == 0) {
		res[numPoints] = 0.0f; // Needed as Lagrange factor for GEMV with inverse covariance matrix of samples (needed for Kriging)
	}
}



// Calculates residuals of samples and an unconditional realization. Uses bilinear interpolation based on the sample's position in grid
__global__ void residualsSimple_3f(float* res, float *srcdata, float *uncond_grid, float3 *indices, int nx, int ny, int nz, int numPoints, float mu) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numPoints) {
		
		// Trilinear interpolation
		float x = indices[id].x; 
		float y = indices[id].y;
		float z = indices[id].z;
		int row = floor(y); // y index of upper neighbour pixel
		int col = floor(x); // x index of lower neighbour pixel
		int zdim = floor(z); // z index of lower neighbour pixel
		x = (float)x - col; // Weight of right neighbour or 1 - weight of left neighbour
		y = (float)y - row; // Weight of lower neighbour or 1 - weight of upper neighbour
		z = (float)z - zdim;
		
		// Special cases
		if (col > nx-1) {
			x = 0.0f;col = nx-1;
		}
		else if (col < 0) {
			x = 0.0f;col=0;
		}
		if (row > nx-1) {
			y = 0.0f;row = nx-y;
		}	
		else if (row < 0) {
			y = 0.0f;row=0;
		}
		if (zdim > nz-1) {
			z = 0.0f;zdim = nz-1;
		}
		else if (zdim < 0) {
			z = 0.0f;zdim = 0;
		}

		float c00 = (1-y)*uncond_grid[zdim*nx*ny + row*nx + col]         +   y*uncond_grid[zdim*nx*ny + row*nx + col+1];
		float c10 = (1-y)*uncond_grid[(zdim+1)*nx*ny + row*nx + col]     +   y*uncond_grid[(zdim+1)*nx*ny + row*nx + col+1];
		float c01 = (1-y)*uncond_grid[zdim*nx*ny + (row+1)*nx + col]     +   y*uncond_grid[zdim*nx*ny + (row+1)*nx + col+1];
		float c11 = (1-y)*uncond_grid[(zdim+1)*nx*ny + (row+1)*nx + col] +   y*uncond_grid[(zdim+1)*nx*ny + (row+1)*nx + col+1];

		c00 = (1-z) * c00 + z*c10;
		c01 = (1-z) * c01 + z*c11;

		res[id] = srcdata[id] - ((1-x)*c00 + x*c01);
		
		/*res[id] = srcdata[id] - ((1-y) * ((1-x) * uncond_grid[row * nx + col] + x * uncond_grid[row * nx + col + 1]) + 
								  y * ((1-x) * uncond_grid[(row+1) * nx + col] + x * uncond_grid[(row+1) * nx + col + 1]));*/
	}		
}








/// Kriging prediction at a regular grid with given samples for conditioning
#ifndef BLOCK_SIZE_KRIGE1
#define BLOCK_SIZE_KRIGE1 256
#endif


__global__ void krigingKernel_3f(float *prediction, float3 *srcXY, float xmin, float dx, float ymin, float dy, float zmin, float dz, float *y, int model, float range, float sill, float nugget, int numSrc, int nx, int ny, int nz)
{	
	int bx = blockIdx.x;
    int tx = threadIdx.x;

	float sum=0.0;
	float yr_x, yr_y, yr_z;
	
	__shared__ float qs[BLOCK_SIZE_KRIGE1];
	__shared__ float Xs[BLOCK_SIZE_KRIGE1][3];

    if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){
		yr_z = zmin + dz * (int)((bx*BLOCK_SIZE_KRIGE1 + tx)/(nx*ny));
		yr_y = ymin + dy * (ny-1-   (int)(((bx*BLOCK_SIZE_KRIGE1 + tx)%(nx*ny))/nx));
		yr_x = xmin + dx * ((bx*BLOCK_SIZE_KRIGE1 + tx)%nx);

	}
	__syncthreads();
	for (int b=0;b<numSrc;b+=BLOCK_SIZE_KRIGE1){
		
		if ((b+tx)<numSrc){         
			Xs[tx][0]=srcXY[(tx+b)].x;
			Xs[tx][1]=srcXY[(tx+b)].y;
			Xs[tx][2]=srcXY[(tx+b)].z;
			qs[tx]=y[tx+b];
		}
		__syncthreads();
		if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){	
			for (int i=0;i<BLOCK_SIZE_KRIGE1;++i){
				if ((b+i)<numSrc){
					switch (model) {
					case EXP:
						sum += covExpKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i];  
						break;
					case GAU:
						sum += covGauKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case SPH:
						sum += covSphKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case MAT3:
						sum += covMat3Kernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case MAT5:
						sum += covMat5Kernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					}
				}
			}
		}
		__syncthreads();      
	}
	if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz) prediction[bx*BLOCK_SIZE_KRIGE1 + tx] = sum + y[numSrc];	
}





__global__ void krigingAnisKernel_3f(float *prediction, float3 *srcXY, float xmin, float dx, float ymin, float dy, float zmin, float dz, float *y, int model, float range, float sill, float nugget, float alpha, float beta, float theta, float afac1, float afac2, int numSrc, int nx, int ny, int nz)
{	
	int bx = blockIdx.x;
    int tx = threadIdx.x;

	float sum=0.0;
	float yr_x, yr_y, yr_z;
	
	__shared__ float qs[BLOCK_SIZE_KRIGE1];
	__shared__ float Xs[BLOCK_SIZE_KRIGE1][3];

    if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){
		yr_z = zmin + dz * (int)((bx*BLOCK_SIZE_KRIGE1 + tx)/(nx*ny));
		yr_y = ymin + dy * (ny-1-   (int)(((bx*BLOCK_SIZE_KRIGE1 + tx)%(nx*ny))/nx));
		yr_x = xmin + dx * ((bx*BLOCK_SIZE_KRIGE1 + tx)%nx);

	}
	__syncthreads();
	for (int b=0;b<numSrc;b+=BLOCK_SIZE_KRIGE1){
		
		if ((b+tx)<numSrc){         
			Xs[tx][0]=srcXY[(tx+b)].x;
			Xs[tx][1]=srcXY[(tx+b)].y;
			Xs[tx][2]=srcXY[(tx+b)].z;
			qs[tx]=y[tx+b];
		}
		__syncthreads();
		if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){	
			for (int i=0;i<BLOCK_SIZE_KRIGE1;++i){
				if ((b+i)<numSrc){
					switch (model) {
					case EXP:
						sum += covExpAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i];  
						break;
					case GAU:
						sum += covGauAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case SPH:
						sum += covSphAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case MAT3:
						sum += covMat3AnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case MAT5:
						sum += covMat5AnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					}
				}
			}
		}
		__syncthreads();      
	}
	if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz) prediction[bx*BLOCK_SIZE_KRIGE1 + tx] = sum + y[numSrc];	
}






__global__ void krigingSimpleKernel_3f(float *prediction, float3 *srcXY, float xmin, float dx, float ymin, float dy, float zmin, float dz, float *y, int model, float range, float sill, float nugget, int numSrc, int nx, int ny, int nz, float mean)
{	
	int bx = blockIdx.x;
    int tx = threadIdx.x;

	float sum=0.0;
	float yr_x, yr_y, yr_z;
	
	__shared__ float qs[BLOCK_SIZE_KRIGE1];
	__shared__ float Xs[BLOCK_SIZE_KRIGE1][3];

    if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){
		yr_z = zmin + dz * (int)((bx*BLOCK_SIZE_KRIGE1 + tx)/(nx*ny));
		yr_y = ymin + dy * (ny-1-   (int)(((bx*BLOCK_SIZE_KRIGE1 + tx)%(nx*ny))/nx));
		yr_x = xmin + dx * ((bx*BLOCK_SIZE_KRIGE1 + tx)%nx);

	}
	__syncthreads();
	for (int b=0;b<numSrc;b+=BLOCK_SIZE_KRIGE1){
		
		if ((b+tx)<numSrc){         
			Xs[tx][0]=srcXY[(tx+b)].x;
			Xs[tx][1]=srcXY[(tx+b)].y;
			Xs[tx][2]=srcXY[(tx+b)].z;
			qs[tx]=y[tx+b];
		}
		__syncthreads();
		if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){	
			for (int i=0;i<BLOCK_SIZE_KRIGE1;++i){
				if ((b+i)<numSrc){
					switch (model) {
					case EXP:
						sum += covExpKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i];  
						break;
					case GAU:
						sum += covGauKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case SPH:
						sum += covSphKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case MAT3:
						sum += covMat3Kernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					case MAT5:
						sum += covMat5Kernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget) *qs[i]; 
						break;
					}
				}
			}
		}
		__syncthreads();      
	}
	if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz) prediction[bx*BLOCK_SIZE_KRIGE1 + tx] = sum + mean;	
}






__global__ void krigingSimpleAnisKernel_3f(float *prediction, float3 *srcXY, float xmin, float dx, float ymin, float dy, float zmin, float dz, float *y, int model, float range, float sill, float nugget, float alpha, float beta, float theta, float afac1, float afac2, int numSrc, int nx, int ny, int nz, float mean)
{	
	int bx = blockIdx.x;
    int tx = threadIdx.x;

	float sum=0.0;
	float yr_x, yr_y, yr_z;
	
	__shared__ float qs[BLOCK_SIZE_KRIGE1];
	__shared__ float Xs[BLOCK_SIZE_KRIGE1][3];

    if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){
		yr_z = zmin + dz * (int)((bx*BLOCK_SIZE_KRIGE1 + tx)/(nx*ny));
		yr_y = ymin + dy * (ny-1-   (int)(((bx*BLOCK_SIZE_KRIGE1 + tx)%(nx*ny))/nx));
		yr_x = xmin + dx * ((bx*BLOCK_SIZE_KRIGE1 + tx)%nx);

	}
	__syncthreads();
	for (int b=0;b<numSrc;b+=BLOCK_SIZE_KRIGE1){
		
		if ((b+tx)<numSrc){         
			Xs[tx][0]=srcXY[(tx+b)].x;
			Xs[tx][1]=srcXY[(tx+b)].y;
			Xs[tx][2]=srcXY[(tx+b)].z;
			qs[tx]=y[tx+b];
		}
		__syncthreads();
		if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz){	
			for (int i=0;i<BLOCK_SIZE_KRIGE1;++i){
				if ((b+i)<numSrc){
					switch (model) {
					case EXP:
						sum += covExpAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i];  
						break;
					case GAU:
						sum += covGauAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case SPH:
						sum += covSphAnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case MAT3:
						sum += covMat3AnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					case MAT5:
						sum += covMat5AnisKernel_3f(yr_x,yr_y,yr_z,Xs[i][0],Xs[i][1],Xs[i][2],sill,range,nugget,alpha,beta,theta,afac1,afac2) *qs[i]; 
						break;
					}
				}
			}
		}
		__syncthreads();      
	}
	if ((bx*BLOCK_SIZE_KRIGE1 + tx)<nx*ny*nz) prediction[bx*BLOCK_SIZE_KRIGE1 + tx] = sum + mean;	
}









/*******************************************************************************************
** UNCONDITIONAL SIMULATION  ***************************************************************
********************************************************************************************/

// global variables for unconditional simulation. These data are needed in the preprocessing as well as in generating realizations
struct uncond_state_3f {
	hipfftComplex *d_cov; // d_cov is the result of the preprocessing ans is needed for each realozation
	int nx,ny,nz,n,m,o;
	float xmin,xmax,ymin,ymax,zmin,zmax,dx,dy,dz;
	int blockSize,numBlocks;
	dim3 blockSize2, numBlocks2;
	hipfftHandle plan1;
	dim3 blockSize3d;
	dim3 blockCount3d;
	dim3 blockSize1d;
	dim3 blockCount1d;
} uncond_global_3f;


#ifdef __cplusplus
extern "C" {
#endif


void EXPORT unconditionalSimInit_3f(float *p_xmin, float *p_xmax, int *p_nx, float *p_ymin, float *p_ymax, int *p_ny, float *p_zmin, float *p_zmax, int *p_nz, float *p_sill, float *p_range, float *p_nugget, int *p_covmodel, float *p_anis, int *do_check, int *ret_code) {
	*ret_code = OK;
	hipError_t cudaStatus;
	
	uncond_global_3f.nx= *p_nx; // Number of cols
	uncond_global_3f.ny= *p_ny; // Number of rows
	uncond_global_3f.nz= *p_nz; // Number of z dims
	uncond_global_3f.n= 2*uncond_global_3f.nx; // Number of cols
	uncond_global_3f.m= 2*uncond_global_3f.ny; // Number of rows
	uncond_global_3f.o= 2*uncond_global_3f.nz; // Number of zdims
	//uncond_global_3f.n = ceil2(2*uncond_global_3f.nx); /// 
	//uncond_global_3f.m = ceil2(2*uncond_global_3f.ny); /// 
	//uncond_global_3f.o = ceil2(2*uncond_global_3f.nz); /// 
	uncond_global_3f.dx = (*p_xmax - *p_xmin) / (uncond_global_3f.nx-1);
	uncond_global_3f.dy = (*p_ymax - *p_ymin) / (uncond_global_3f.ny-1);
	uncond_global_3f.dz = (*p_zmax - *p_zmin) / (uncond_global_3f.nz-1);

	// 1d cuda grid
	uncond_global_3f.blockSize1d = dim3(256);
	uncond_global_3f.blockCount1d = dim3(uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o / uncond_global_3f.blockSize1d.x);
	if (uncond_global_3f.n * uncond_global_3f.m * uncond_global_3f.o % uncond_global_3f.blockSize1d.x  != 0) ++uncond_global_3f.blockCount1d.x;
	
	// 3d cuda grid
	uncond_global_3f.blockSize3d = dim3(8,8,4);
	uncond_global_3f.blockCount3d = dim3(uncond_global_3f.n / uncond_global_3f.blockSize3d.x, uncond_global_3f.m / uncond_global_3f.blockSize3d.y, uncond_global_3f.o / uncond_global_3f.blockSize3d.z);
	if (uncond_global_3f.n % uncond_global_3f.blockSize3d.x != 0) ++uncond_global_3f.blockCount3d.x;
	if (uncond_global_3f.m % uncond_global_3f.blockSize3d.y != 0) ++uncond_global_3f.blockCount3d.y;
	if (uncond_global_3f.o % uncond_global_3f.blockSize3d.z != 0) ++uncond_global_3f.blockCount3d.z;
	
	hipfftPlan3d(&uncond_global_3f.plan1, uncond_global_3f.m, uncond_global_3f.n, uncond_global_3f.o, HIPFFT_C2C); 

	
	// build grid (ROW MAJOR)
	// 3d grid:
	// z in [0,o-1] y in [0,n-1], x in [0,m-1]
	// --> f(x,y,z) = f[z*(n*m) + y*  ???????????????????�


	/*hipfftComplex *h_grid_c = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.m*uncond_global_3f.n*uncond_global_3f.o);
	for (int i=0; i<uncond_global_3f.n; ++i) { // i =  col index
		for (int j=0; j<uncond_global_3f.m; ++j) { // j = row index 
			h_grid_c[j*uncond_global_3f.n+i].x = *p_xmin + (i+1) * uncond_global_3f.dx; 
			h_grid_c[j*uncond_global_3f.n+i].y = *p_ymin + (j+1) * uncond_global_3f.dy;  
		}
	}*/
	
	float3 *h_grid_c = (float3 *)malloc(sizeof(float3)*uncond_global_3f.m*uncond_global_3f.n*uncond_global_3f.o);
	for (int k=0; k<uncond_global_3f.o; ++k) {
		for (int i=0; i<uncond_global_3f.n; ++i) { // i =  col index
			for (int j=0; j<uncond_global_3f.m; ++j) { // j = row index 
				h_grid_c[k*uncond_global_3f.n*uncond_global_3f.m + j*uncond_global_3f.n + i].x =  *p_xmin + (i+1) * uncond_global_3f.dx; 
				//h_grid_c[k*uncond_global_3f.n*uncond_global_3f.m + j*uncond_global_3f.n + i].y =  *p_ymin + (j+1) * uncond_global_3f.dy;
				h_grid_c[k*uncond_global_3f.n*uncond_global_3f.m + j*uncond_global_3f.n + i].y =  *p_ymin + (uncond_global_3f.m-1-j)* uncond_global_3f.dy;
				h_grid_c[k*uncond_global_3f.n*uncond_global_3f.m + j*uncond_global_3f.n + i].z =  *p_zmin + (k+1) * uncond_global_3f.dz; 
			}
		}
	}


	
	float xc = *p_xmin + (uncond_global_3f.dx*uncond_global_3f.n)/2;
	float yc = *p_ymin +(uncond_global_3f.dy*uncond_global_3f.m)/2;
	float zc = *p_ymin +(uncond_global_3f.dz*uncond_global_3f.o)/2;
	float sill = *p_sill;
	float range = *p_range;
	float nugget = *p_nugget;

	bool isotropic = (p_anis[3] == 1.0 && p_anis[4] == 1.0);
	float alpha = (90.0 - p_anis[0]) * (PI / 180.0);
	float beta = -1.0 * p_anis[1] * (PI / 180.0);
	float theta = p_anis[2] * (PI / 180.0);
	float afac1 = 1/p_anis[3];
	float afac2 = 1/p_anis[4];

	
	float3 *d_grid;
	
	// Array for grid
	cudaStatus = hipMalloc((void**)&d_grid,sizeof(float3)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);
	// Array for cov grid
	cudaStatus = hipMalloc((void**)&uncond_global_3f.d_cov,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);

	// Sample covariance and generate "trick" grid
	hipfftComplex *d_trick_grid_c;
	cudaStatus = hipMalloc((void**)&d_trick_grid_c,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);
	
	// copy grid to GPU
	cudaStatus = hipMemcpy(d_grid,h_grid_c, uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	if (isotropic) {
		sampleCovKernel_3f<<<uncond_global_3f.blockCount3d, uncond_global_3f.blockSize3d>>>(d_trick_grid_c, d_grid, uncond_global_3f.d_cov, xc, yc, zc,*p_covmodel, sill, range,nugget,uncond_global_3f.n,uncond_global_3f.m,uncond_global_3f.o);		
	}
	else {
		sampleCovAnisKernel_3f<<<uncond_global_3f.blockCount3d, uncond_global_3f.blockSize3d>>>(d_trick_grid_c, d_grid, uncond_global_3f.d_cov, xc, yc, zc,*p_covmodel, sill, range, nugget, alpha, beta, theta, afac1, afac2,uncond_global_3f.n,uncond_global_3f.m,uncond_global_3f.o);
	}
	free(h_grid_c);
	hipFree(d_grid);
	 
//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE COV MATRIX******* ///////
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,uncond_global_3f.d_cov,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\sampleCov.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif

//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE TRICK GRID ******* /////// 
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,d_trick_grid_c,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\trickgrid.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif
//


	// Execute 3d FFT of covariance grid in order to get the spectral representation 
	hipfftExecC2C(uncond_global_3f.plan1, uncond_global_3f.d_cov, uncond_global_3f.d_cov, HIPFFT_FORWARD); // in place fft forward


//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE FFT( COV GRID) ******* /////// 
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,uncond_global_3f.d_cov,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\fftSampleCov.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif
//
	hipfftExecC2C(uncond_global_3f.plan1, d_trick_grid_c, d_trick_grid_c, HIPFFT_FORWARD); // in place fft forward

//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE FFT( TRICK GRID) ******* /////// 
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,d_trick_grid_c,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\fftTrickGrid.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif
	
	// Multiply fft of "trick" grid with n*m
	multKernel_3f<<<uncond_global_3f.blockCount1d, uncond_global_3f.blockSize1d>>>(d_trick_grid_c, uncond_global_3f.n, uncond_global_3f.m, uncond_global_3f.o);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching multKernel_3f!\n", cudaStatus);	

//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE FFT( TRICK GRID) ******* /////// 
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,d_trick_grid_c,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\fftTrickGridTimesNM.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif


	// Devide spectral covariance grid by "trick" grid
	divideSpectrumKernel_3f<<<uncond_global_3f.blockCount1d, uncond_global_3f.blockSize1d>>>(uncond_global_3f.d_cov, d_trick_grid_c);	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching divideSpectrumKernel_f!\n", cudaStatus);	
	hipFree(d_trick_grid_c);

	
//#ifdef DEBUG 
//	{
//		/// ****** TEST AUSGABE FFT( COV GRID) / FFT(TRICKGRID)*N*M ******* /////// 
//		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m);
//		cudaStatus = hipMemcpy(h_cov,uncond_global_3f.d_cov,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m,hipMemcpyDeviceToHost);
//		writeCSVMatrix("C:\\fft\\fftSampleCovByTrickGridNM.csv",h_cov,uncond_global_3f.m,uncond_global_3f.n);
//		free(h_cov);
//	}
//#endif
//


	// Copy to host and check for negative real parts
	if (*do_check) {
		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);
		cudaStatus = hipMemcpy(h_cov,uncond_global_3f.d_cov,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o,hipMemcpyDeviceToHost);
		for (int i=0; i<uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o; ++i) {
			if (h_cov[i].x < 0.0) {
				*ret_code = ERROR_NEGATIVE_COV_VALUES; 
				free(h_cov);
				hipFree(uncond_global_3f.d_cov);
				hipfftDestroy(uncond_global_3f.plan1);
				return;
			}	
		}
		free(h_cov);
	}

	// Compute sqrt of cov grid
	sqrtKernel_3f<<<uncond_global_3f.blockCount1d, uncond_global_3f.blockSize1d>>>(uncond_global_3f.d_cov);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching sqrtKernel_f\n", cudaStatus);

}

// Generates unconditional realizations
// p_out = output array of size nx*ny*k * sizeof(float)
// p_k = Number of realizations
// ret_code = return code: 0=ok
void EXPORT unconditionalSimRealizations_3f(float *p_out,  int *p_k, int *ret_code)
{
	*ret_code = OK;
	hipError_t cudaStatus;

	int k = *p_k;

	float *d_rand; // device random numbers
	hiprandGenerator_t curandGen;
	hipfftComplex *d_fftrand;
	hipfftComplex* d_amp;
	float* d_out;

	hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(curandGen,(unsigned long long)(time(NULL)));	

	cudaStatus = hipMalloc((void**)&d_rand,sizeof(float)*uncond_global_3f.m*uncond_global_3f.n*uncond_global_3f.o); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);

	cudaStatus = hipMalloc((void**)&d_fftrand,sizeof(hipfftComplex) * uncond_global_3f.n * uncond_global_3f.m * uncond_global_3f.o); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);

	dim3 blockSize3dhalf  = dim3(8,8,4);
	dim3 blockCount3dhalf = dim3(uncond_global_3f.nx/blockSize3dhalf.x,uncond_global_3f.ny/blockSize3dhalf.y,uncond_global_3f.nz/blockSize3dhalf.z);
	if (uncond_global_3f.nx % blockSize3dhalf.x != 0) ++blockCount3dhalf.x;
	if (uncond_global_3f.ny % blockSize3dhalf.y != 0) ++blockCount3dhalf.y;
	if (uncond_global_3f.nz % blockSize3dhalf.z != 0) ++blockCount3dhalf.z;

	for(int l = 0; l<k; ++l) {
				
		// Generate Random Numbers
		hiprandGenerateNormal(curandGen,d_rand,uncond_global_3f.m*uncond_global_3f.n*uncond_global_3f.o,0.0,1.0);
		
		// Convert real random numbers to complex numbers
		realToComplexKernel_3f<<< uncond_global_3f.blockCount1d, uncond_global_3f.blockSize1d>>>(d_fftrand, d_rand, uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) printf("hipDeviceSynchronize returned error code %d after launching realToComplexKernel_f!\n", cudaStatus);	

		// Compute 2D FFT of random numbers
		hipfftExecC2C(uncond_global_3f.plan1, d_fftrand, d_fftrand, HIPFFT_FORWARD); // in place fft forward

		if(l==0) hipMalloc((void**)&d_amp,sizeof(hipfftComplex)*uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o);
		elementProduct_3f<<<uncond_global_3f.blockCount1d, uncond_global_3f.blockSize1d>>>(d_amp, uncond_global_3f.d_cov, d_fftrand, uncond_global_3f.m*uncond_global_3f.n*uncond_global_3f.o);  
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching elementProduct_f!\n", cudaStatus);

		hipfftExecC2C(uncond_global_3f.plan1, d_amp, d_amp, HIPFFT_BACKWARD); // in place fft inverse for simulation		
		if(l==0) hipMalloc((void**)&d_out,sizeof(float)*uncond_global_3f.nx*uncond_global_3f.ny*uncond_global_3f.nz);
		
		
		ReDiv_3f<<<blockCount3dhalf, blockSize3dhalf>>>(d_out, d_amp, std::sqrt((float)(uncond_global_3f.n*uncond_global_3f.m*uncond_global_3f.o)), uncond_global_3f.nx, uncond_global_3f.ny, uncond_global_3f.nz);
		cudaStatus = hipDeviceSynchronize();	
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching ReDiv_3f!\n", cudaStatus);
		}
		hipMemcpy((p_out + l*(uncond_global_3f.nx*uncond_global_3f.ny*uncond_global_3f.nz)),d_out,sizeof(float)*uncond_global_3f.nx*uncond_global_3f.ny*uncond_global_3f.nz,hipMemcpyDeviceToHost);
	}

	hipFree(d_rand);
	hipFree(d_fftrand);
	hipFree(d_amp);
	hipFree(d_out);
	hiprandDestroyGenerator(curandGen);
}


void EXPORT unconditionalSimRelease_3f(int *ret_code) {
	*ret_code = OK;
	hipFree(uncond_global_3f.d_cov);
	hipfftDestroy(uncond_global_3f.plan1);
}


#ifdef __cplusplus
}
#endif
















/*******************************************************************************************
** CONDITIONAL SIMULATION  ***************************************************************
********************************************************************************************/


// global variables for conditional simulation that are needed both, for initialization as well as for generating realizations
struct cond_state_3f {
	hipfftComplex *d_cov; 
	int nx,ny,nz,n,m,o;
	float xmin,xmax,zmin,zmax,ymin,ymax,dx,dy,dz;
	float range, sill, nugget;
	float alpha,beta,theta,afac1,afac2;
	bool isotropic;
	int blockSize,numBlocks;
	dim3 blockSize2, numBlocks2;
	hipfftHandle plan1;
	dim3 blockSize3d;
	dim3 blockCount3d;
	dim3 blockSize1d;
	dim3 blockCount1d;
	dim3 blockSizeSamples;
	dim3 blockCountSamples;
	dim3 blockSizeSamplesPlus1;
	dim3 blockCountSamplesPlus1;
	// Variables for conditioning
	int numSrc; // Number of sample observation
	float3 *d_samplexy; // coordinates of samples
	float3 *d_sampleindices; // Corresponding grid indices in subpixel accuracy
	float *d_sampledata; // data values of samples
	//float *d_covinv; // inverse covariance matrix of samples
	float *d_uncond;
	int covmodel;
	int k;
	float mu; // known mean for simple kriging
	int krige_method;
} cond_global_3f;









#ifdef __cplusplus
extern "C" {
#endif





void EXPORT conditionalSimInit_3f(float *p_xmin, float *p_xmax, int *p_nx, float *p_ymin, float *p_ymax, 
								  int *p_ny, float *p_zmin, float *p_zmax, int *p_nz, float *p_sill, float *p_range, 
								  float *p_nugget, float *p_srcXY,  float *p_srcData, int *p_numSrc, int *p_covmodel, 
								  float *p_anis, int *do_check, int *krige_method, float *mu, int *ret_code) {
	*ret_code = OK;
	hipError_t cudaStatus;
	cublasInit();

	cond_global_3f.nx= *p_nx; // Number of cols
	cond_global_3f.ny= *p_ny; // Number of rows
	cond_global_3f.nz= *p_nz; // Number of rows
	cond_global_3f.n= 2*cond_global_3f.nx; // Number of cols
	cond_global_3f.m= 2*cond_global_3f.ny; // Number of rows
	cond_global_3f.o= 2*cond_global_3f.ny; // Number of rows
	cond_global_3f.dx = (*p_xmax - *p_xmin) / (cond_global_3f.nx - 1);
	cond_global_3f.dy = (*p_ymax - *p_ymin) / (cond_global_3f.ny - 1);
	cond_global_3f.dz = (*p_zmax - *p_zmin) / (cond_global_3f.nz - 1);
	cond_global_3f.numSrc = *p_numSrc;
	cond_global_3f.xmin = *p_xmin;
	cond_global_3f.xmax = *p_xmax;
	cond_global_3f.ymin = *p_ymin;
	cond_global_3f.ymax = *p_ymax;
	cond_global_3f.zmin = *p_zmin;
	cond_global_3f.zmax = *p_zmax;
	cond_global_3f.range = *p_range;
	cond_global_3f.sill = *p_sill;
	cond_global_3f.nugget = *p_nugget;
	cond_global_3f.covmodel = *p_covmodel;
	cond_global_3f.krige_method = *krige_method;
	if (cond_global_3f.krige_method == SIMPLE)
		cond_global_3f.mu = *mu;
	else cond_global_3f.mu = 0;

	cond_global_3f.isotropic = (p_anis[3] == 1.0 && p_anis[4] == 1.0);
	cond_global_3f.alpha = (90.0 - p_anis[0]) * (PI / 180.0);
	cond_global_3f.beta = -1.0 * p_anis[1] * (PI / 180.0);
	cond_global_3f.theta = p_anis[2] * (PI / 180.0);
	cond_global_3f.afac1 = 1/p_anis[3];
	cond_global_3f.afac2 = 1/p_anis[4];



	// 1d cuda grid
	cond_global_3f.blockSize1d = dim3(256);
	cond_global_3f.blockCount1d = dim3(cond_global_3f.n*cond_global_3f.m*cond_global_3f.o / cond_global_3f.blockSize1d.x);
	if (cond_global_3f.n * cond_global_3f.m * cond_global_3f.o % cond_global_3f.blockSize1d.x  != 0) ++cond_global_3f.blockCount1d.x;
	
	// 3d cuda grid
	cond_global_3f.blockSize3d = dim3(8,8,4);
	cond_global_3f.blockCount3d = dim3(cond_global_3f.n / cond_global_3f.blockSize3d.x, cond_global_3f.m / cond_global_3f.blockSize3d.y, cond_global_3f.o / cond_global_3f.blockSize3d.z);
	if (cond_global_3f.n % cond_global_3f.blockSize3d.x != 0) ++cond_global_3f.blockCount3d.x;
	if (cond_global_3f.m % cond_global_3f.blockSize3d.y != 0) ++cond_global_3f.blockCount3d.y;
	if (cond_global_3f.o % cond_global_3f.blockSize3d.z != 0) ++cond_global_3f.blockCount3d.z;
	
	hipfftPlan3d(&cond_global_3f.plan1, cond_global_3f.m, cond_global_3f.n, cond_global_3f.o, HIPFFT_C2C); 

	
	// build grid (ROW MAJOR)

	// 1d cuda grid for samples
	cond_global_3f.blockSizeSamples = dim3(256);
	cond_global_3f.blockCountSamples = dim3(cond_global_3f.numSrc / cond_global_3f.blockSizeSamples.x);
	if (cond_global_3f.numSrc % cond_global_3f.blockSizeSamples.x !=0) ++cond_global_3f.blockCountSamples.x;

	
	

	float3 *h_grid_c = (float3 *)malloc(sizeof(float3)*cond_global_3f.m*cond_global_3f.n*cond_global_3f.o);
	for (int k=0; k<cond_global_3f.o; ++k) {
		for (int i=0; i<cond_global_3f.n; ++i) { // i =  col index
			for (int j=0; j<cond_global_3f.m; ++j) { // j = row index 
				h_grid_c[k*cond_global_3f.n*cond_global_3f.m + j*cond_global_3f.n + i].x =  *p_xmin + (i+1) * cond_global_3f.dx; 
				//h_grid_c[k*cond_global_3f.n*cond_global_3f.m + j*cond_global_3f.n + i].y =  *p_ymin + (j+1) * cond_global_3f.dy;
				h_grid_c[k*cond_global_3f.n*cond_global_3f.m + j*cond_global_3f.n + i].y =  *p_ymin + (cond_global_3f.m-1-j)* cond_global_3f.dy;
				h_grid_c[k*cond_global_3f.n*cond_global_3f.m + j*cond_global_3f.n + i].z =  *p_zmin + (k+1) * cond_global_3f.dz; 
			}
		}
	}

	float xc = *p_xmin + (cond_global_3f.dx*cond_global_3f.n)/2;
	float yc = *p_ymin +(cond_global_3f.dy*cond_global_3f.m)/2;
	float zc = *p_ymin +(cond_global_3f.dz*cond_global_3f.o)/2;
	
	float3 *d_grid;
	
	// Allocate grid and cov arrays on GPU
	cudaStatus = hipMalloc((void**)&d_grid,sizeof(float3)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);
	cudaStatus = hipMalloc((void**)&cond_global_3f.d_cov,sizeof(hipfftComplex)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);

	// Sample covariance and generate "trick" grid
	hipfftComplex *d_trick_grid_c;
	cudaStatus = hipMalloc((void**)&d_trick_grid_c,sizeof(hipfftComplex)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);
	
	// copy grid to GPU
	cudaStatus = hipMemcpy(d_grid,h_grid_c, cond_global_3f.n*cond_global_3f.m*cond_global_3f.o*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	if (cond_global_3f.isotropic) {
		sampleCovKernel_3f<<<cond_global_3f.blockCount3d, cond_global_3f.blockSize3d>>>(d_trick_grid_c, d_grid, cond_global_3f.d_cov, xc, yc, zc,*p_covmodel, cond_global_3f.sill, cond_global_3f.range,cond_global_3f.nugget,cond_global_3f.n,cond_global_3f.m,cond_global_3f.o);		
	}
	else {
		sampleCovAnisKernel_3f<<<cond_global_3f.blockCount3d, cond_global_3f.blockSize3d>>>(d_trick_grid_c, d_grid, cond_global_3f.d_cov, xc, yc, zc,*p_covmodel, cond_global_3f.sill, cond_global_3f.range, cond_global_3f.nugget, cond_global_3f.alpha, cond_global_3f.beta, cond_global_3f.theta, cond_global_3f.afac1, cond_global_3f.afac2,cond_global_3f.n,cond_global_3f.m,cond_global_3f.o);
	}
	free(h_grid_c);
	hipFree(d_grid);



	// Compute spectral representation of cov and "trick" grid
	hipfftExecC2C(cond_global_3f.plan1, cond_global_3f.d_cov, cond_global_3f.d_cov, HIPFFT_FORWARD); // in place fft forward
	hipfftExecC2C(cond_global_3f.plan1, d_trick_grid_c, d_trick_grid_c, HIPFFT_FORWARD); // in place fft forwar


	// Multiplication of fft(trick_grid) with n*m	
	multKernel_3f<<<cond_global_3f.blockCount1d, cond_global_3f.blockSize1d>>>(d_trick_grid_c, cond_global_3f.n, cond_global_3f.m, cond_global_3f.o);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching multKernel_3f!\n", cudaStatus);	

	// Devide spectral cov grid by fft of "trick" grid
	divideSpectrumKernel_3f<<<cond_global_3f.blockCount1d, cond_global_3f.blockSize1d>>>(cond_global_3f.d_cov, d_trick_grid_c);	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching divideSpectrumKernel_f!\n", cudaStatus);	
	hipFree(d_trick_grid_c);

	// Copy to host and check for negative real parts
	if (*do_check) {
		hipfftComplex *h_cov = (hipfftComplex*)malloc(sizeof(hipfftComplex)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);
		cudaStatus = hipMemcpy(h_cov,cond_global_3f.d_cov,sizeof(hipfftComplex)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o,hipMemcpyDeviceToHost);
		for (int i=0; i<cond_global_3f.n*cond_global_3f.m*cond_global_3f.o; ++i) {
			if (h_cov[i].x < 0.0) {
				*ret_code = ERROR_NEGATIVE_COV_VALUES; 
				free(h_cov);
				hipFree(cond_global_3f.d_cov);
				hipfftDestroy(cond_global_3f.plan1);
				return;
			}	
		}
		free(h_cov);
	}



	// Compute sqrt of spectral cov grid
	sqrtKernel_3f<<<cond_global_3f.blockCount1d, cond_global_3f.blockSize1d>>>(cond_global_3f.d_cov);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching sqrtKernel_f\n", cudaStatus);

	// Copy samples to gpu
	cudaStatus = hipMalloc((void**)&cond_global_3f.d_samplexy,sizeof(float3)* cond_global_3f.numSrc); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);
	cudaStatus = hipMalloc((void**)&cond_global_3f.d_sampleindices,sizeof(float3)*cond_global_3f.numSrc); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);
	cudaStatus = hipMalloc((void**)&cond_global_3f.d_sampledata,sizeof(float)*cond_global_3f.numSrc); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);
	hipMemcpy(cond_global_3f.d_samplexy,p_srcXY,sizeof(float3)* cond_global_3f.numSrc,hipMemcpyHostToDevice);
	hipMemcpy(cond_global_3f.d_sampledata,p_srcData,sizeof(float)*cond_global_3f.numSrc,hipMemcpyHostToDevice);
		

	// Overlay samples to grid and save resulting subpixel grind indices
	overlay_3f<<<cond_global_3f.blockCountSamples,cond_global_3f.blockSizeSamples>>>(cond_global_3f.d_sampleindices,cond_global_3f.d_samplexy,*p_xmin,cond_global_3f.dx,*p_ymax,cond_global_3f.dy,cond_global_3f.zmin, cond_global_3f.dz, cond_global_3f.numSrc);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching overlay_3f!\n", cudaStatus);	
	// Warning: It is not checked, whether sample points truly lie inside the grid's boundaries. This may lead to illegal memory access			

	/* TEST OUTPUT ON HOST */
	/*float2 *h_indices = (float2*)malloc(sizeof(float2)*cond_global_3f.numSrc);
	hipMemcpy(h_indices,cond_global_3f.d_sampleindices,sizeof(float2)*cond_global_3f.numSrc,hipMemcpyDeviceToHost);
	for (int i=0;i<cond_global_3f.numSrc;++i) {
		printf("(%.2f,%.2f) -> (%.2f,%.2f)\n",p_srcXY[2*i],p_srcXY[2*i+1],h_indices[i].x, h_indices[i].y);
	}
	free(h_indices);*/
}




// Generates Unconditional Realizations and the residuals of all samples to all realizations 
// p_out = output matrix of residuals, col means number of realization, row represents a sample point
// p_k = Number of realizations
// ret_code = return code: 0=ok
void EXPORT conditionalSimUncondResiduals_3f(float *p_out, int *p_k, int *ret_code) {
	*ret_code = OK;
	hipError_t cudaStatus;
	cond_global_3f.k = *p_k;
	
	float *d_rand; // Device Random Numbers
	hiprandGenerator_t curandGen;
	hipfftComplex *d_fftrand;
	hipfftComplex* d_amp;	
	float *d_residuals; // residuals of samples and underlying unconditional realization
	
	hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(curandGen,(unsigned long long)(time(NULL)));	
	
	cudaStatus = hipMalloc((void**)&d_rand,sizeof(float)*cond_global_3f.m*cond_global_3f.n*cond_global_3f.o); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);
	cudaStatus = hipMalloc((void**)&d_fftrand,sizeof(hipfftComplex) * cond_global_3f.n * cond_global_3f.m * cond_global_3f.o); 
	if (cudaStatus != hipSuccess)  printf("hipMalloc returned error code %d\n", cudaStatus);
	hipMalloc((void**)&d_amp,sizeof(hipfftComplex)*cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);
	hipMalloc((void**)&cond_global_3f.d_uncond,sizeof(float)*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz * cond_global_3f.k);
	
	
	if (cond_global_3f.krige_method == ORDINARY) {
		hipMalloc((void**)&d_residuals,sizeof(float)* (cond_global_3f.numSrc + 1));
	}
	else if (cond_global_3f.krige_method == SIMPLE) {
		hipMalloc((void**)&d_residuals,sizeof(float)* cond_global_3f.numSrc);
	}
		
	for(int l=0; l<cond_global_3f.k; ++l) {
			
		
		hiprandGenerateNormal(curandGen,d_rand,cond_global_3f.m*cond_global_3f.n*cond_global_3f.o,0.0,1.0);	
		realToComplexKernel_3f<<< cond_global_3f.blockCount1d, cond_global_3f.blockSize1d>>>(d_fftrand, d_rand, cond_global_3f.n*cond_global_3f.m*cond_global_3f.o);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching realToComplexKernel_f!\n", cudaStatus);	
		hipfftExecC2C(cond_global_3f.plan1, d_fftrand, d_fftrand, HIPFFT_FORWARD); // in place fft forward
		cudaStatus = hipDeviceSynchronize();
		
		elementProduct_3f<<<cond_global_3f.blockCount1d, cond_global_3f.blockSize1d>>>(d_amp, cond_global_3f.d_cov, d_fftrand, cond_global_3f.m*cond_global_3f.n*cond_global_3f.o);
    
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching elementProduct_f!\n", cudaStatus);

		hipfftExecC2C(cond_global_3f.plan1, d_amp, d_amp, HIPFFT_BACKWARD); // in place fft inverse for simulation
	  
		dim3 blockSize3dhalf  = dim3(8,8,4);
		dim3 blockCount3dhalf = dim3(cond_global_3f.nx/blockSize3dhalf.x,cond_global_3f.ny/blockSize3dhalf.y,cond_global_3f.nz/blockSize3dhalf.z);
		if (cond_global_3f.nx % blockSize3dhalf.x != 0) ++blockCount3dhalf.x;
		if (cond_global_3f.ny % blockSize3dhalf.y != 0) ++blockCount3dhalf.y;
		if (cond_global_3f.nz % blockSize3dhalf.z != 0) ++blockCount3dhalf.z;
		ReDiv_3f<<<blockCount3dhalf, blockSize3dhalf>>>(cond_global_3f.d_uncond + l*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz, d_amp, std::sqrt((float)(cond_global_3f.n*cond_global_3f.m*cond_global_3f.o)), cond_global_3f.nx, cond_global_3f.ny, cond_global_3f.nz);
		cudaStatus = hipDeviceSynchronize();	
		if (cudaStatus != hipSuccess) printf("hipDeviceSynchronize returned error code %d after launching ReDiv_3f!\n", cudaStatus);
		
		// d_uncond is now a unconditional realization 
		// Compute residuals between samples and d_uncond
		if (cond_global_3f.krige_method == ORDINARY) {
			residualsOrdinary_3f<<<cond_global_3f.blockCountSamples,cond_global_3f.blockSizeSamples>>>(d_residuals,cond_global_3f.d_sampledata,cond_global_3f.d_uncond+l*(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz),cond_global_3f.d_sampleindices,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz,cond_global_3f.numSrc);
		}
		else if (cond_global_3f.krige_method == SIMPLE) {
			residualsSimple_3f<<<cond_global_3f.blockCountSamples,cond_global_3f.blockSizeSamples>>>(d_residuals,cond_global_3f.d_sampledata,cond_global_3f.d_uncond+l*(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz),cond_global_3f.d_sampleindices,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz,cond_global_3f.numSrc, cond_global_3f.mu);
		}


		cudaStatus = hipDeviceSynchronize();	
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching residuals!\n", cudaStatus);
	

		// Copy residuals to R, col major...
		if (cond_global_3f.krige_method == ORDINARY) {
			hipMemcpy((p_out + l*(cond_global_3f.numSrc + 1)),d_residuals,sizeof(float)* (cond_global_3f.numSrc + 1),hipMemcpyDeviceToHost);	
		}
		else if (cond_global_3f.krige_method == SIMPLE) {
			hipMemcpy(p_out + l*cond_global_3f.numSrc,d_residuals,sizeof(float) * cond_global_3f.numSrc,hipMemcpyDeviceToHost);	
		}
	}
	hiprandDestroyGenerator(curandGen);
	
	hipFree(d_rand);
	hipFree(d_fftrand);
	hipFree(d_amp);
	hipFree(d_residuals);
}


void EXPORT conditionalSimKrigeResiduals_3f(float *p_out, float *p_y, int *ret_code)
{
	*ret_code = OK;
	hipError_t cudaStatus = hipSuccess;
	
	float *d_y; // result vector from solving the kriging equation system
	float *d_respred; // interpolated residuals
	hipMalloc((void**)&d_y, sizeof(float) * (cond_global_3f.numSrc + 1));
	hipMalloc((void**)&d_respred, sizeof(float) * cond_global_3f.nx * cond_global_3f.ny  * cond_global_3f.nz);
	
	dim3 blockSizeKrige = dim3(BLOCK_SIZE_KRIGE1);
	dim3 blockCntKrige = dim3((cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz) / blockSizeKrige.x);
	if ((cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz) % blockSizeKrige.x != 0) ++blockCntKrige.x;
	
	dim3 blockSizeCond = dim3(256);
	dim3 blockCntCond = dim3(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz/ blockSizeCond.x);
	if (cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz % blockSizeCond.x != 0) ++blockSizeCond.x;

	for(int l = 0; l<cond_global_3f.k; ++l) {
						
		
		hipMemcpy(d_y, p_y + l*(cond_global_3f.numSrc + 1), sizeof(float) * (cond_global_3f.numSrc + 1),hipMemcpyHostToDevice);		
		
		// Kriging prediction
		if (cond_global_3f.isotropic)
			krigingKernel_3f<<<blockCntKrige, blockSizeKrige>>>(d_respred,cond_global_3f.d_samplexy,cond_global_3f.xmin,cond_global_3f.dx,cond_global_3f.ymin,cond_global_3f.dy,cond_global_3f.zmin,cond_global_3f.dz,d_y,cond_global_3f.covmodel,cond_global_3f.range,cond_global_3f.sill,cond_global_3f.nugget,cond_global_3f.numSrc,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz);
		else 	
			krigingAnisKernel_3f<<<blockCntKrige, blockSizeKrige>>>(d_respred,cond_global_3f.d_samplexy,cond_global_3f.xmin,cond_global_3f.dx,cond_global_3f.ymin,cond_global_3f.dy,cond_global_3f.zmin,cond_global_3f.dz,d_y,cond_global_3f.covmodel,cond_global_3f.range,cond_global_3f.sill,cond_global_3f.nugget, cond_global_3f.alpha, cond_global_3f.beta, cond_global_3f.theta, cond_global_3f.afac1, cond_global_3f.afac2, cond_global_3f.numSrc,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz);
		

		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching krigingExpKernel_3f!\n", cudaStatus);
		
		// Add result to unconditional realization
		addResSim_3f<<<blockCntCond,blockSizeCond>>>(d_respred, cond_global_3f.d_uncond + l*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz, cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz);
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching addResSim_f!\n", cudaStatus);
		
		// Write Result to R
		hipMemcpy((p_out + l*(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz)),d_respred,sizeof(float)*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz,hipMemcpyDeviceToHost);		
		
	}
	hipFree(d_y);
	hipFree(d_respred);
}




void EXPORT conditionalSimSimpleKrigeResiduals_3f(float *p_out, float *p_y, int *ret_code)
{
	*ret_code = OK;
	hipError_t cudaStatus = hipSuccess;
	
	float *d_y; // result vector from solving the kriging equation system
	float *d_respred; // interpolated residuals
	hipMalloc((void**)&d_y, sizeof(float) * cond_global_3f.numSrc); // not + 1, no lagrange multiplicator in simple kriging
	hipMalloc((void**)&d_respred, sizeof(float) * cond_global_3f.nx * cond_global_3f.ny *cond_global_3f.nz);
	
	dim3 blockSizeKrige = dim3(BLOCK_SIZE_KRIGE1);
	dim3 blockCntKrige = dim3((cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz) / blockSizeKrige.x);
	if ((cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz) % blockSizeKrige.x != 0) ++blockCntKrige.x;
	
	dim3 blockSizeCond = dim3(256);
	dim3 blockCntCond = dim3(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz/ blockSizeCond.x);
	if (cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz % blockSizeCond.x != 0) ++blockSizeCond.x;

	for(int l = 0; l<cond_global_3f.k; ++l) {
						
		hipMemcpy(d_y, p_y + l*cond_global_3f.numSrc, sizeof(float) * cond_global_3f.numSrc,hipMemcpyHostToDevice);	// not + 1, no lagrange multiplicator in simple kriging		
		// Kriging prediction
		if (cond_global_3f.isotropic)
			krigingSimpleKernel_3f<<<blockCntKrige, blockSizeKrige>>>(d_respred,cond_global_3f.d_samplexy,cond_global_3f.xmin,cond_global_3f.dx,cond_global_3f.ymin,cond_global_3f.dy,cond_global_3f.zmin,cond_global_3f.dz,d_y,cond_global_3f.covmodel,cond_global_3f.range,cond_global_3f.sill,cond_global_3f.nugget,cond_global_3f.numSrc,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz,cond_global_3f.mu);
		else 	
			krigingSimpleAnisKernel_3f<<<blockCntKrige, blockSizeKrige>>>(d_respred,cond_global_3f.d_samplexy,cond_global_3f.xmin,cond_global_3f.dx,cond_global_3f.ymin,cond_global_3f.dy,cond_global_3f.zmin,cond_global_3f.dz,d_y,cond_global_3f.covmodel,cond_global_3f.range,cond_global_3f.sill,cond_global_3f.nugget, cond_global_3f.alpha, cond_global_3f.beta, cond_global_3f.theta, cond_global_3f.afac1, cond_global_3f.afac2, cond_global_3f.numSrc,cond_global_3f.nx,cond_global_3f.ny,cond_global_3f.nz,cond_global_3f.mu);
		

		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching krigingExpKernel_3f!\n", cudaStatus);
		
		// Add result to unconditional realization
		addResSim_3f<<<blockCntCond,blockSizeCond>>>(d_respred, cond_global_3f.d_uncond + l*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz, cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz);
		if (cudaStatus != hipSuccess)  printf("hipDeviceSynchronize returned error code %d after launching addResSim_f!\n", cudaStatus);
		
		// Write Result to R
		hipMemcpy((p_out + l*(cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz)),d_respred,sizeof(float)*cond_global_3f.nx*cond_global_3f.ny*cond_global_3f.nz,hipMemcpyDeviceToHost);		
		
	}
	hipFree(d_y);
	hipFree(d_respred);
}





void EXPORT conditionalSimRelease_3f(int *ret_code) {
	*ret_code = OK;
	hipfftDestroy(cond_global_3f.plan1);
	hipFree(cond_global_3f.d_samplexy);
	hipFree(cond_global_3f.d_sampledata);
	hipFree(cond_global_3f.d_sampleindices);
	hipFree(cond_global_3f.d_cov);
	hipFree(cond_global_3f.d_uncond);
}




#ifdef __cplusplus
}
#endif










